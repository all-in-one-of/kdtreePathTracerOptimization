#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include <glm/gtc/matrix_inverse.hpp>
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#include <algorithm>
#include <stdlib.h>

#include <random>
#include <vector>
#include <stack>
#include <string>
#include <fstream>
#include <iostream>
#include <iomanip>

#include <glm/gtx/intersect.hpp>

#include "KDnode.h"
#include "KDtree.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
    int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}













static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static PathSegment * dev_paths_cache = NULL;
static ShadeableIntersection * dev_intersections = NULL;

static const int STACK_SIZE = 2000;


struct is_zero_bounce
{
    __host__ __device__
        bool operator()(const PathSegment p)
    {
        return (p.remainingBounces == 0);
    }
};

struct by_material_id
{
    const PathSegment a;
    by_material_id(PathSegment _a) : a(_a) {}
    __host__ __device__
        int operator()(const PathSegment& x, const PathSegment& y) const
    {
        //x.
        return x.color.r + y.color.r;
    }
};

__host__ __device__ bool operator<(const PathSegment &lhs, const PathSegment &rhs)
{
    return lhs.materialIdHit < rhs.materialIdHit;
}

__host__ __device__ bool operator<(const ShadeableIntersection &lhs, const ShadeableIntersection &rhs)
{
    return lhs.materialId < rhs.materialId;
}


struct NodeStack{
    KDN::NodeBare* node;
    float tmin;
    float tmax;
    glm::vec3 origin;
};




// ------------------------------------------------------------------------
// --------------------------- KD TREE UTILITIES --------------------------
// ------------------------------------------------------------------------

std::vector<KDN::Triangle*> getTrianglesFromFile(const char* path)
{
    std::vector<KDN::Triangle*>triangles;

    string line;
    ifstream file(path);

    if (file.is_open())
    {
        while (getline(file, line))
        {
            float x1 = atof(line.c_str());
            getline(file, line); float y1 = atof(line.c_str());
            getline(file, line); float z1 = atof(line.c_str());
            getline(file, line); float x2 = atof(line.c_str());
            getline(file, line); float y2 = atof(line.c_str());
            getline(file, line); float z2 = atof(line.c_str());
            getline(file, line); float x3 = atof(line.c_str());
            getline(file, line); float y3 = atof(line.c_str());
            getline(file, line); float z3 = atof(line.c_str());

            KDN::Triangle* t = new KDN::Triangle(x1, y1, z1,
                                                 x2, y2, z2,
                                                 x3, y3, z3);
            triangles.push_back(t);
        }
    }
    return triangles;
}


/*
// fast AABB intersection
__host__ __device__ //__global__
void intersectAABB(Ray r, KDN::BoundingBox b, float& dist, bool& result)
{

    glm::vec3 invdir(1.0f / r.direction.x,
                     1.0f / r.direction.y,
                     1.0f / r.direction.z);

    float v1 = (b.mins[0] - r.origin.x)*invdir.x;
    float v2 = (b.maxs[0] - r.origin.x)*invdir.x;
    float v3 = (b.mins[1] - r.origin.y)*invdir.y;
    float v4 = (b.maxs[1] - r.origin.y)*invdir.y;
    float v5 = (b.mins[2] - r.origin.z)*invdir.z;
    float v6 = (b.maxs[2] - r.origin.z)*invdir.z;

    float dmin = max(max(min(v1, v2), min(v3, v4)), min(v5, v6));
    float dmax = min(min(max(v1, v2), max(v3, v4)), max(v5, v6));

    if (dmax < 0)
    {
        dist = dmax;
        result = false;
        return;
    }
    if (dmin > dmax)
    {
        dist = dmax;
        result = false;
        return;
    }
    dist = dmin;
    result = true;
    return;
}
*/

/*
void getKDnodes(KDN::KDnode* root, vector<KDN::KDnode*>& nodes)
{
    if (root != NULL)
    {
        nodes.push_back(root);
        getKDnodes(root->left, nodes);
        getKDnodes(root->right, nodes);
    }
}

void getKDnodesLoop(KDN::KDnode* root, vector<KDN::KDnode*>& nodes)
{
    KDN::KDnode* currNode = root;
    while (true)
    {
        if (currNode == NULL)
            break;

        if (currNode->left != NULL && currNode->left->visited != true)
            currNode = currNode->left;
        else if (currNode->right != NULL && currNode->right->visited != true)
            currNode = currNode->right;
        else if (currNode->visited == false)
        {
            //std::cout << "NODE LOOP: " << currNode << std::endl;
            nodes.push_back(currNode);
            currNode->visited = true;
        }
        else
            currNode = currNode->parent;
    }

    // reset visited to false
    currNode = root;
    while (true)
    {
        if (currNode == NULL)
            break;

        if (currNode->left != NULL && currNode->left->visited != false)
            currNode = currNode->left;
        else if (currNode->right != NULL && currNode->right->visited != false)
            currNode = currNode->right;
        else if (currNode->visited == true)
            currNode->visited = false;
        else
            currNode = currNode->parent;
    }
}


void getKDnodesLoopDeref(KDN::KDnode* root, vector<KDN::KDnode>& nodes)
{
    KDN::KDnode* currNode = root;

    while (true)
    {
        if (currNode == NULL)
            break;

        if (currNode->left != NULL && currNode->left->visited != true)
            currNode = currNode->left;
        else if (currNode->right != NULL && currNode->right->visited != true)
            currNode = currNode->right;
        else if (currNode->visited == false)
        {
            //std::cout << "NODE LOOP: " << currNode << std::endl;
            nodes.push_back(currNode[0]);
            currNode->visited = true;
        }
        else
            currNode = currNode->parent;
    }

    // reset visited to false
    currNode = root;
    while (true)
    {
        if (currNode == NULL)
            break;

        if (currNode->left != NULL && currNode->left->visited != false)
            currNode = currNode->left;
        else if (currNode->right != NULL && currNode->right->visited != false)
            currNode = currNode->right;
        else if (currNode->visited == true)
            currNode->visited = false;
        else
            currNode = currNode->parent;
    }
}



vector<int> cacheTriangles(KDN::KDnode* nodes, int numNodes, vector<KDN::Triangle>& newTriangles)
{

    int triCount = 0;
    vector<int> offsets;

    if (numNodes == 0)
        return offsets;

    for (int i = 0; i < numNodes; i++)
    {
        int numTriangles = nodes[i].triangles.size();
        if (numTriangles > 0)
        {
            // update node triangle lookup
            nodes[i].triIdStart = triCount;
            nodes[i].triIdSize = numTriangles;

            triCount += numTriangles;
            offsets.push_back(triCount);

            for (int j = 0; j < numTriangles; j++)
            {
                newTriangles.push_back(nodes[i].triangles[j][0]);
            }
        }

        std::cout << "node: " << nodes[i].ID << " numtris: " << numTriangles << std::endl;
    }


    return offsets;
}


vector<int> cacheTriangles(std::vector<KDN::KDnode*> nodes, vector<KDN::Triangle>& newTriangles)
{

    int triCount = 0;
    vector<int> offsets;

    if (nodes.size() == 0)
        return offsets;

    for (int i = 0; i < nodes.size(); i++)
    {
        int numTriangles = nodes[i]->triangles.size();
        if (numTriangles > 0)
        {
            // update node triangle lookup
            nodes[i]->triIdStart = triCount;
            nodes[i]->triIdSize = numTriangles;

            triCount += numTriangles;
            offsets.push_back(triCount);

            for (int j = 0; j < numTriangles; j++)
            {
                newTriangles.push_back(nodes[i]->triangles[j][0]);
            }
        }

        std::cout << "node: " << nodes[i]->ID << " numtris: " << numTriangles << std::endl;
    }


    return offsets;
}

vector<int> cacheTriangles(std::vector<KDN::KDnode> nodes, vector<KDN::Triangle>& newTriangles)
{

    int triCount = 0;
    vector<int> offsets;

    if (nodes.size() == 0)
        return offsets;

    for (int i = 0; i < nodes.size(); i++)
    {
        int numTriangles = nodes[i].triangles.size();
        if (numTriangles > 0)
        {
            // update node triangle lookup
            nodes[i].triIdStart = triCount;
            nodes[i].triIdSize = numTriangles;

            triCount += numTriangles;
            offsets.push_back(triCount);

            for (int j = 0; j < numTriangles; j++)
            {
                newTriangles.push_back(nodes[i].triangles[j][0]);
            }
        }

        std::cout << "node: " << nodes[i].ID << " numtris: " << numTriangles << std::endl;
    }


    return offsets;
}

void deleteTree(KDN::KDnode* root)
{
    if (root != NULL)
    {
        deleteTree(root->left);
        deleteTree(root->right);
        //delete root;

        if (root->left != NULL)
            root->left = NULL;
        if (root->right != NULL)
            root->right = NULL;

        delete root;
        root = NULL;
    }
}

bool nodeComparator(const void* a, const void* b)
{
    int ida = (*(KDN::KDnode*)a).ID;
    int idb = (*(KDN::KDnode*)b).ID;

    if (ida <= idb)
        return true;
    else if (ida > idb)
        return false;
}


std::vector<KDN::Triangle*> getTrianglesFromScene(Scene* scene)
{
    vector<KDN::Triangle*> triangles;

    int iterator = 0;

    int pidxo1 = 0;
    int pidxo2 = 0;
    int pidxo3 = 0;

    for (int i = 0; i < scene->obj_numshapes; i++)
    {
        for (int j = iterator; j < iterator + scene->obj_polyoffsets[i]; j += 3)
        {
            pidxo1 = 3 * scene->obj_polysidxflat[j];
            pidxo2 = 3 * scene->obj_polysidxflat[j + 1];
            pidxo3 = 3 * scene->obj_polysidxflat[j + 2];

            KDN::Triangle* t = new KDN::Triangle(
                scene->obj_verts[pidxo1],
                scene->obj_verts[pidxo1 + 1],
                scene->obj_verts[pidxo1 + 2],
                scene->obj_verts[pidxo2],
                scene->obj_verts[pidxo2 + 1],
                scene->obj_verts[pidxo2 + 2],
                scene->obj_verts[pidxo3],
                scene->obj_verts[pidxo3 + 1],
                scene->obj_verts[pidxo3 + 2],
                scene->obj_norms[pidxo1],
                scene->obj_norms[pidxo1 + 1],
                scene->obj_norms[pidxo1 + 2],
                scene->obj_norms[pidxo2],
                scene->obj_norms[pidxo2 + 1],
                scene->obj_norms[pidxo2 + 2],
                scene->obj_norms[pidxo3],
                scene->obj_norms[pidxo3 + 1],
                scene->obj_norms[pidxo3 + 2]);

            triangles.push_back(t);
        }

        iterator += scene->obj_polyoffsets[i];
    }

    return triangles;
}
*/


// ------------------------------------------------------------------------
// ------------------------ KD TREE UTILITIES END -------------------------
// ------------------------------------------------------------------------













int obj_numshapes = 0;
int* obj_numpolyverts = NULL;
//int** obj_polysidx = NULL;
float* obj_verts = NULL;
float* obj_norms = NULL;
float* obj_texts = NULL;
int* obj_polyoffsets = NULL;
int* obj_polysidxflat = NULL;
float* obj_polysbboxes = NULL;

// KD DATA
//KDN::KDnode* kd_nodes = NULL;
//KDN::Triangle* kd_triangles = NULL;
KDN::NodeBare* kd_nodesBare = NULL;
KDN::TriBare* kd_trianglesBare = NULL;
static int numNodes = 0;
static int numTriangles = 0;
//

/*
float* obj_RGB = NULL;
float* obj_SPECEX = NULL;
float* obj_SPECRGB = NULL;
float* obj_REFL = NULL;
float* obj_REFR = NULL;
float* obj_REFRIOR = NULL;
*/
int* obj_materialOffsets = NULL;

void pathtraceInit(Scene *scene, bool enablekd) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
    hipMalloc(&dev_paths_cache, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
 

    // objloader part
    if (scene->hasObj)
    {
        if (enablekd == false)
        {
            hipMalloc((void**)&obj_numpolyverts, scene->obj_numshapes * sizeof(int));
            hipMalloc((void**)&obj_polyoffsets, scene->obj_numshapes * sizeof(int));
            hipMalloc((void**)&obj_polysidxflat, scene->polyidxcount * sizeof(int));
            hipMalloc((void**)&obj_verts, scene->objmesh->attrib.vertices.size()* sizeof(float));
            hipMalloc((void**)&obj_norms, scene->objmesh->attrib.normals.size()* sizeof(float));
            hipMalloc((void**)&obj_texts, scene->objmesh->attrib.texcoords.size()* sizeof(float));
            hipMalloc((void**)&obj_polysbboxes, scene->obj_numshapes * 6 * sizeof(float));
        }
        hipMalloc((void**)&obj_materialOffsets, scene->obj_numshapes * sizeof(int));


        // ------------------------------------------------------------------
        // KD DATA PART
        // ------------------------------------------------------------------
        /*
        // This section assembles a KD tree and flattens out the entire tree
        // as an array of KDnodes, an array of Triangles and an array of 
        // indices with the triangle offsets so that we can traverse the tree
        // non recursively.
        std::vector<KDN::Triangle*> triangles = getTrianglesFromScene(scene);

        KDtree* KDT = new KDtree(triangles);
        KDT->rootNode->updateBbox();
        KDT->split(3);

        // Accessing kd nodes and triangles as a flat structure
        // This is to help recursion removal for CUDA
        // THANK YOU NVIDIA for this...
        vector<KDN::KDnode*> nodes;
        getKDnodes(KDT->rootNode, nodes);
        std::sort(nodes.begin(), nodes.end(), nodeComparator);

        // flattened triangles data
        vector<KDN::Triangle> newTriangles;
        vector<int> offsets = cacheTriangles(nodes, newTriangles);
        numTriangles = newTriangles.size();

        vector<KDN::KDnode*> nodesLoop;
        getKDnodesLoop(KDT->rootNode, nodesLoop);
        std::sort(nodesLoop.begin(), nodesLoop.end());

        vector<KDN::KDnode> nodesLoopDeref;
        getKDnodesLoopDeref(KDT->rootNode, nodesLoopDeref);
        std::sort(nodesLoopDeref.begin(), nodesLoopDeref.end());

        
        // flattened nodes data
        numNodes = nodesLoopDeref.size();
        KDN::KDnode* newNodes = new KDN::KDnode[numNodes];
        memcpy(newNodes, nodesLoopDeref.data(), sizeof(KDN::KDnode)*numNodes);
        */

        if (enablekd == true)
        {
            /*
            printf("size of:\n\ttriangle: %d\n\ttribare: %d\n\tnode: %d\n\tnodebare: %d\n", 
                   sizeof(KDN::Triangle),
                   sizeof(KDN::TriBare),
                   sizeof(KDN::KDnode),
                   sizeof(KDN::NodeBare));
            */

            //printf("TOTAL NUM NODES = %d\n", scene->numNodes);

            // allocate KD data on the device
            //hipMalloc((void**)&kd_nodes, scene->numNodes * sizeof(KDN::KDnode));
            //hipMalloc((void**)&kd_triangles, scene->numTriangles * sizeof(KDN::Triangle));
            //printf("before allocating\n");
            hipMalloc((void**)&kd_nodesBare, scene->numNodes * sizeof(KDN::NodeBare));
            hipMalloc((void**)&kd_trianglesBare, scene->numTriangles * sizeof(KDN::TriBare));
            //printf("after allocating\n");
            
            //hipMemcpy(kd_nodes, scene->newNodes, scene->numNodes * sizeof(KDN::KDnode), hipMemcpyHostToDevice);
            //hipMemcpy(kd_triangles, scene->newTriangles, scene->numTriangles * sizeof(KDN::Triangle), hipMemcpyHostToDevice);
            //printf("before copying\n");
            hipMemcpy(kd_nodesBare, scene->newNodesBare, scene->numNodes * sizeof(KDN::NodeBare), hipMemcpyHostToDevice);
            hipMemcpy(kd_trianglesBare, scene->newTrianglesBare, scene->numTriangles * sizeof(KDN::TriBare), hipMemcpyHostToDevice);
            //printf("after copying\n");
        }
        else
        {
            hipMemcpy(obj_numpolyverts, scene->obj_numpolyverts, scene->obj_numshapes * sizeof(int), hipMemcpyHostToDevice);
            //printf("step 2\n");
            hipMemcpy(obj_polyoffsets, scene->obj_polyoffsets, scene->obj_numshapes * sizeof(int), hipMemcpyHostToDevice);
            //printf("step 3\n"); 
            hipMemcpy(obj_polysidxflat, scene->obj_polysidxflat, scene->polyidxcount * sizeof(int), hipMemcpyHostToDevice);
            //printf("step 4\n"); 
            hipMemcpy(obj_verts, scene->obj_verts, scene->objmesh->attrib.vertices.size()* sizeof(float), hipMemcpyHostToDevice);
            //printf("step 5\n"); 
            hipMemcpy(obj_norms, scene->obj_norms, scene->objmesh->attrib.normals.size()* sizeof(float), hipMemcpyHostToDevice);
            //printf("step 6\n"); 
            hipMemcpy(obj_texts, scene->obj_texts, scene->objmesh->attrib.texcoords.size()* sizeof(float), hipMemcpyHostToDevice);
            //printf("step 7\n"); 
            hipMemcpy(obj_polysbboxes, scene->obj_bboxes, scene->obj_numshapes * 6 * sizeof(float), hipMemcpyHostToDevice);
            //printf("step 8\n"); 
        }
        hipMemcpy(obj_materialOffsets, scene->obj_materialOffsets, scene->obj_numshapes * sizeof(int), hipMemcpyHostToDevice);
        //printf("step 9\n");
    }
    // shading 
    /*
    hipMemcpy(obj_RGB, scene->obj_RGB, scene->obj_numshapes * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(obj_SPECEX, scene->obj_SPECEX, scene->obj_numshapes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(obj_SPECRGB, scene->obj_SPECRGB, scene->obj_numshapes * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(obj_REFL, scene->obj_REFL, scene->obj_numshapes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(obj_REFR, scene->obj_REFR, scene->obj_numshapes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(obj_REFRIOR, scene->obj_REFRIOR, scene->obj_numshapes * sizeof(float), hipMemcpyHostToDevice);
    */
 
    checkCUDAError("pathtraceInit");
}






























void pathtraceFree(Scene *scene, bool enablekd) {
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_paths_cache);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created

    // objloader part
    if (scene->hasObj)
    {
        if (enablekd == false)
        {
            hipFree(obj_numpolyverts);
            hipFree(obj_polyoffsets);
            hipFree(obj_polysidxflat);
            hipFree(obj_verts);
            hipFree(obj_norms);
            hipFree(obj_texts);
        }
        hipFree(obj_materialOffsets);

        if (enablekd == true)
        {
            // KD DATA
            //hipFree(kd_nodes);
            //hipFree(kd_triangles);
            hipFree(kd_nodesBare);
            hipFree(kd_trianglesBare);
        }
    }
    // shading
    /*
    hipFree(obj_RGB);
    hipFree(obj_SPECEX);
    hipFree(obj_SPECRGB);
    hipFree(obj_REFL);
    hipFree(obj_REFR);
    hipFree(obj_REFRIOR);
    */
    

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments, float focalLength, float dofAngle, bool antialias)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment & segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);
        segment.ray.isinside = false;

        // store initial index
        //segment.initialidx = index;

        
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
            );

        // TODO: implement antialiasing by jittering the ray
        thrust::default_random_engine rng(utilhash(iter));
        thrust::uniform_real_distribution<float> unitDistrib(0, 1);
        if (antialias)
        {
            float jitterscale = 0.002;
            //thrust::uniform_real_distribution<float> unitDistrib(0, 1);

            bool fast = true;
            if (fast)
            {
                // use cheap jitter
                glm::vec3 v3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
                v3 = glm::normalize(v3);
                segment.ray.direction += v3*jitterscale;
                segment.ray.direction = glm::normalize(segment.ray.direction);
            }
            else
            {
                // use uniform spherical distribution
                float u = cos(PI * (float)unitDistrib(rng));
                float u2 = u*u;
                float sqrt1minusu2 = sqrt(1 - u2);
                float theta = 2 * PI * (float)unitDistrib(rng);
                glm::vec3  v3(sqrt1minusu2 * cos(theta),
                    sqrt1minusu2 * sin(theta),
                    u);
                segment.ray.direction += v3*jitterscale;
            }
        }




        // depth of field
        //thrust::uniform_real_distribution<float> unitDistrib01(-1, 1);
        
        // use uniform spherical distribution
        float u = cos(PI * (float)unitDistrib(rng));
        float u2 = u*u;
        float sqrt1minusu2 = sqrt(1 - u2);
        float theta = 2 * PI * (float)unitDistrib(rng);
        glm::vec3  v3(sqrt1minusu2 * cos(theta),
            sqrt1minusu2 * sin(theta),
            u);
        v3 = glm::normalize(v3);
        

        
        glm::vec3 center = cam.position + 8.0f * segment.ray.direction;
        //center -= cam.position;

        float R1 = (float)unitDistrib(rng);
        float R2 = (float)unitDistrib(rng);

        /*
        float angleVal = 0.25*PI;
        glm::vec3 randRotAngle(glm::cos(PI*R1 * glm::sin(angleVal*0.5f*R2)),
                               glm::sin(PI*R1 * glm::sin(angleVal*0.5f*R2)),
                               glm::cos(angleVal*0.5f*R2));
        */
        
        //v3 = glm::normalize(glm::cross(v3, segment.ray.direction));

        glm::vec3 front = glm::normalize(cam.lookAt);
        glm::vec3 up = glm::normalize(cam.up);
        glm::vec3 right = glm::normalize(cam.right);
        glm::quat Q1;
        //glm::vec3 a = glm::normalize(glm::cross(segment.ray.direction, cam.right));
        float randangle = (float)unitDistrib(rng) * PI * dofAngle;
        Q1.w = cosf(randangle / 2.0f);
        Q1.x = v3.x * sinf(randangle / 2.0f);
        Q1.y = v3.y * sinf(randangle / 2.0f);
        Q1.z = v3.z * sinf(randangle / 2.0f);
       
        glm::vec3 randrot = glm::rotate(Q1, segment.ray.direction);
        //center += cam.position;
        
        
        
        segment.ray.origin = segment.ray.origin + segment.ray.direction * focalLength - randrot*focalLength;
        segment.ray.direction = randrot;


        segment.ray.direction = glm::normalize(segment.ray.direction);


        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}







// pathTraceOneBounce handles ray intersections, generate intersections for shading, 
// and scatter new ray. You might want to call scatterRay from interactions.h
__global__ void pathTraceOneBounce(
    int depth
    , int iter
    , int num_paths
    , PathSegment * pathSegments
    , Geom * geoms
    , int geoms_size
    , Material * materials
    , int material_size
    , ShadeableIntersection * intersections
    , float softness
    , int obj_numshapes
    , int* obj_numpolyverts
    , float* obj_verts
    , float* obj_norms
    , float* obj_texts
    , int* obj_polyoffsets
    , int* obj_polysidxflat
    , float* obj_polysbboxes
    , int polyidxcount
    /*,
    float* obj_RGB
    , float* obj_SPECEX
    , float* obj_SPECRGB
    , float* obj_REFL
    , float* obj_REFR
    , float* obj_REFRIOR
    */
    , int* obj_materialOffsets
    , int hasobj
    )
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        //path_index = pathSegments[path_index].pixelIndex;
        PathSegment pathSegment = pathSegments[path_index];
        //printf("\nO1");
        if (pathSegments[path_index].remainingBounces>0)
        {
            float t;
            glm::vec3 intersect_point;
            glm::vec3 normal;
            float t_min = FLT_MAX;
            int hit_geom_index = -1;
            bool outside = true;

            glm::vec3 tmp_intersect;
            glm::vec3 tmp_normal;

            glm::vec3 hit;
            glm::vec3 norm;
            glm::vec3 bary;
            glm::vec3 v1;
            glm::vec3 v2;
            glm::vec3 v3;
            glm::vec3 n1;
            glm::vec3 n2;
            glm::vec3 n3;
            int pidxo1 = 0;
            int pidxo2 = 0;
            int pidxo3 = 0;
            bool intersected = false;
            bool obj_intersect = false;
            // naive parse through global geoms
            //printf("\nO2");

            int objMaterialIdx = -1;
            for (int i = 0; i < geoms_size; i++)
            {
                Geom & geom = geoms[i];

                if (geom.type == CUBE)
                {
                    t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }
                else if (geom.type == SPHERE)
                {
                    t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }
                // TODO: add more intersection tests here... triangle? metaball? CSG?

                // Compute the minimum t from the intersection tests to determine what
                // scene geometry object was hit first.
                if (t > 0.0f && t_min > t)
                {
                    t_min = t;
                    hit_geom_index = i;
                    intersect_point = tmp_intersect;
                    normal = tmp_normal;
                }
            }

            // start polygon hits
            //t_min = FLT_MAX;
            //for (int i = 0; i < obj_numshapes; i++)
            //    printf("\noffset = %d", obj_polyoffsets[i]);


            //printf("\nO3");
            //printf("\nNUMSHAPES = %d\n", obj_numshapes);
            objMaterialIdx = -1;
            int iterator = 0;
            if (hasobj)
            {
                for (int i = 0; i < obj_numshapes; i++)
                {
                    objMaterialIdx = obj_materialOffsets[i];
                    //printf("\nmaterial index = %d", objMaterialIdx);

                    // check bounding intersection first
                    float T = intersectBbox(pathSegment.ray.origin,
                        pathSegment.ray.direction,
                        glm::vec3(obj_polysbboxes[i] - 0.01, 
                                  obj_polysbboxes[i + 1] - 0.01, 
                                  obj_polysbboxes[i + 2] - 0.01),
                        glm::vec3(obj_polysbboxes[i + 3] + 0.01, 
                                  obj_polysbboxes[i + 4] + 0.01, 
                                  obj_polysbboxes[i + 5] + 0.01));

                    //if (T > -1.0f)
                    {
                        for (int j = iterator; j < iterator + obj_polyoffsets[i]; j += 3)
                        {
                            //printf("\nO5");
                            //int pidx1 = obj_polysidxflat[j];
                            //int pidx2 = obj_polysidxflat[j + 1];
                            //int pidx3 = obj_polysidxflat[j + 2];
                            pidxo1 = 3 * obj_polysidxflat[j];
                            pidxo2 = 3 * obj_polysidxflat[j + 1];
                            pidxo3 = 3 * obj_polysidxflat[j + 2];

                            v1.x = obj_verts[pidxo1];
                            v1.y = obj_verts[pidxo1 + 1];
                            v1.z = obj_verts[pidxo1 + 2];
                            v2.x = obj_verts[pidxo2];
                            v2.y = obj_verts[pidxo2 + 1];
                            v2.z = obj_verts[pidxo2 + 2];
                            v3.x = obj_verts[pidxo3];
                            v3.y = obj_verts[pidxo3 + 1];
                            v3.z = obj_verts[pidxo3 + 2];

                            n1.x = obj_norms[pidxo1];
                            n1.y = obj_norms[pidxo1 + 1];
                            n1.z = obj_norms[pidxo1 + 2];
                            n2.x = obj_norms[pidxo2];
                            n2.y = obj_norms[pidxo2 + 1];
                            n2.z = obj_norms[pidxo2 + 2];
                            n3.x = obj_norms[pidxo3];
                            n3.y = obj_norms[pidxo3 + 1];
                            n3.z = obj_norms[pidxo3 + 2];

                            //printf("\nO6");
                            //bary.x = 0.0f;
                            //bary.y = 0.0f;
                            //bary.z = 0.0f;

                            intersected = false;

                            bary.x = 0.0f; bary.y = 0.0f; bary.z = 0.0f;
                            intersected = glm::intersectRayTriangle(pathSegment.ray.origin,
                                pathSegment.ray.direction,
                                v1, v2, v3, bary);


                            glm::vec3 bary2(bary.x, bary.y, 1.0 - bary.x - bary.y);

                            if (intersected)
                            {
                                //(1 - bary.x - bary.y); bary.x; bary.y
                                //printf("\nO8");
                                hit = pathSegment.ray.origin + pathSegment.ray.direction* bary.z;// (bary2.x * v1 + bary2.y * v2 + bary2.z * v3);
                                norm = glm::normalize((1 - bary.x - bary.y) * n1 + bary.x * n2 + (bary.y) * n3);
                                //norm(glm::normalize(n1));
                                hit += norm*0.0001f;


                                t = glm::distance(pathSegment.ray.origin, hit);

                                if (t > 0.0f && t_min > t)
                                {
                                    t_min = t;
                                    hit_geom_index = obj_materialOffsets[i];
                                    intersect_point = hit;
                                    normal = norm;
                                    tmp_intersect = hit;
                                    tmp_normal = normal;
                                    obj_intersect = true;
                                    intersections[path_index].t = t;
                                }
                            }
                        }
                        iterator += obj_polyoffsets[i];
                        //printf("\nO10");
                    }
                }
            }

                
            
            //printf("\nO11");

            // TODO: scatter the ray, generate intersections for shading
            // feel free to modify the code below

            if (hit_geom_index == -1)
            {
                intersections[path_index].t = -1.0f;
            }
            else
            {
                //The ray hits something
                //intersections[path_index].t = t_min;
                //intersections[path_index].materialId = geoms[hit_geom_index].materialid;
                //intersections[path_index].surfaceNormal = normal;


                // updating rays
                //thrust::default_random_engine rng = makeSeededRandomEngine(iter, depth, depth); // WAY TOO COOL!
                thrust::default_random_engine rng = makeSeededRandomEngine(iter, path_index, depth);

                
                if (obj_intersect)
                {
                    pathSegments[path_index].materialIdHit = objMaterialIdx;

                    scatterRay(pathSegments[path_index].ray,
                        pathSegments[path_index].color,
                        intersect_point,
                        normal,
                        materials[objMaterialIdx],
                        rng,
                        softness);
                }
                else
                {
                    pathSegments[path_index].materialIdHit = geoms[hit_geom_index].materialid;

                    scatterRay(pathSegments[path_index].ray,
                        pathSegments[path_index].color,
                        intersect_point,
                        normal,
                        materials[geoms[hit_geom_index].materialid],
                        rng,
                        softness);
                }
                
                //pathSegments[path_index].ray.direction = calculateRandomDirectionInHemisphere(normal, rng);
                //pathSegments[path_index].ray.origin = intersect_point;


                if (obj_intersect)
                {
                    intersections[path_index].t = t_min;
                    intersections[path_index].materialId = objMaterialIdx; // test material
                    intersections[path_index].surfaceNormal = normal;
                }
                else
                {
                    intersections[path_index].t = t_min;
                    intersections[path_index].materialId = geoms[hit_geom_index].materialid;
                    intersections[path_index].surfaceNormal = normal;
                }
            }
        }
    }
}


// pathTraceOneBounce handles ray intersections, generate intersections for shading, 
// and scatter new ray. You might want to call scatterRay from interactions.h
__global__ void pathTraceOneBounceKDfix(
    int depth
    , int iter
    , int num_paths
    , PathSegment * pathSegments
    , Geom * geoms
    , int geoms_size
    , Material * materials
    , int material_size
    , ShadeableIntersection * intersections
    , float softness
    , KDN::Triangle* triangles
    , int numTriangles
    , KDN::KDnode* nodes
    , int numNodes
    , int* obj_materialOffsets
    , int hasobj
    )
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        //path_index = pathSegments[path_index].pixelIndex;
        PathSegment pathSegment = pathSegments[path_index];
        //printf("\nO1");
        if (pathSegments[path_index].remainingBounces>0)
        {
            float t;
            glm::vec3 intersect_point;
            glm::vec3 normal;
            float t_min = FLT_MAX;
            int hit_geom_index = -1;
            bool outside = true;

            glm::vec3 tmp_intersect;
            glm::vec3 tmp_normal;

            glm::vec3 hit;
            glm::vec3 norm;
            glm::vec3 bary;
            glm::vec3 v1;
            glm::vec3 v2;
            glm::vec3 v3;
            glm::vec3 n1;
            glm::vec3 n2;
            glm::vec3 n3;
            int pidxo1 = 0;
            int pidxo2 = 0;
            int pidxo3 = 0;
            bool intersected = false;
            bool obj_intersect = false;
            // naive parse through global geoms
            //printf("\nO2");

            int objMaterialIdx = -1;
            for (int i = 0; i < geoms_size; i++)
            {
                Geom & geom = geoms[i];

                if (geom.type == CUBE)
                {
                    t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }
                else if (geom.type == SPHERE)
                {
                    t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }

                // Compute the minimum t from the intersection tests to determine what
                // scene geometry object was hit first.
                if (t > 0.0f && t_min > t)
                {
                    t_min = t;
                    hit_geom_index = i;
                    intersect_point = tmp_intersect;
                    normal = tmp_normal;
                }
            }

            objMaterialIdx = -1;
            int iterator = 0;
            if (hasobj)
            {
                //
                //
                if (numNodes != 0)
                {
                    bool nodeIDs[100] = { false };
                    int currID = nodeIDs[nodes[0].ID];
                    float dist = -1.0;

                    // get the root node
                    for (int i = 0; i < numNodes; i++)
                    {
                        if (nodes[i].parentID == -1)
                        {
                            currID = nodes[i].ID;
                            break;
                        }
                    }

                    KDN::KDnode* node = &(nodes[currID]);

                    bool hitGeom = false;
                    float boxdist = -1.0f;
                    bary.z = FLT_MAX;
                    while (true)
                    {
                        if (currID == -1)
                            break;

                        node = &(nodes[currID]);
                        // check if it intersects the bounds
                        //printf("1\n");
                        if (nodeIDs[currID] == true)
                        {
                            nodeIDs[node->ID] = true;
                            nodeIDs[node->leftID] = true;
                            nodeIDs[node->rightID] = true;
                            currID = node->parentID;
                            continue;
                        }
                        else
                        {
                            hitGeom = intersectAABB(pathSegment.ray, node->bbox, dist);
                            //hitGeom = intersectAABBarrays(pathSegment.ray, nodes[currID].mins, nodes[currID].maxs, dist);
                            
                            if (hitGeom == false && node->parentID == -1)
                                break;
                        }
                        //printf("2\n");


                        if (hitGeom == false && dist > bary.z)
                        {
                            nodeIDs[node->ID] = true;
                            nodeIDs[node->leftID] = true;
                            nodeIDs[node->rightID] = true;
                            currID = node->parentID;
                        }
                        else
                        {
                            if (nodes[currID].leftID != -1 && nodeIDs[nodes[currID].leftID] != true)
                                currID = node->leftID;
                            else if (nodes[currID].rightID != -1 && nodeIDs[nodes[currID].rightID] != true)
                                currID = node->rightID;
                            else if (nodeIDs[node->ID] == false)
                            {
                                //printf("NODE LOOP: %d\n", nodes[currID].ID);
                                nodeIDs[node->ID] = true;

                                int size = node->triIdSize;
                                if (size > 0)
                                {
                                    int start = node->triIdStart;
                                    int end = start + size;
                                    for (int i = start; i < end; i++)
                                    {
                                        KDN::Triangle* T = &(triangles[i]);

                                        glm::vec3 v1(T->x1, T->y1, T->z1);
                                        glm::vec3 v2(T->x2, T->y2, T->z2);
                                        glm::vec3 v3(T->x3, T->y3, T->z3);

                                        glm::vec3 n1(T->nx1, T->ny1, T->nz1);
                                        glm::vec3 n2(T->nx2, T->ny2, T->nz2);
                                        glm::vec3 n3(T->nx3, T->ny3, T->nz3);

                                        //bary(0.0f, 0.0f, 0.0f);
                                        bool intersected = glm::intersectRayTriangle(pathSegment.ray.origin,
                                                                                     pathSegment.ray.direction,
                                                                                     v1, v2, v3, bary);

                                        if (intersected)
                                        {
                                            objMaterialIdx = triangles[i].mtlIdx + material_size - 1;
                                            //(1 - bary.x - bary.y); bary.x; bary.y
                                            //printf("material id: %d\n", triangles[i].mtlIdx);
                                            hit = pathSegment.ray.origin + pathSegment.ray.direction* bary.z;// (bary2.x * v1 + bary2.y * v2 + bary2.z * v3);
                                            norm = -glm::normalize((1 - bary.x - bary.y) * n1 + bary.x * n2 + (bary.y) * n3);
                                            //norm(glm::normalize(n1));
                                            hit += norm*0.0001f;


                                            t = glm::distance(pathSegment.ray.origin, hit);

                                            if (t > 0.0f && t_min > t)
                                            {
                                                t_min = t;
                                                hit_geom_index = obj_materialOffsets[T->mtlIdx];
                                                intersect_point = hit;
                                                normal = norm;
                                                tmp_intersect = hit;
                                                tmp_normal = normal;
                                                obj_intersect = true;
                                                intersections[path_index].t = t;
                                            }
                                        }
                                    }
                                }
                            }
                            else
                                currID = node->parentID;
                        }
                    }
                }
            }



            if (hit_geom_index == -1)
            {
                intersections[path_index].t = -1.0f;
            }
            else
            {

                // updating rays
                //thrust::default_random_engine rng = makeSeededRandomEngine(iter, depth, depth); // WAY TOO COOL!
                thrust::default_random_engine rng = makeSeededRandomEngine(iter, path_index, depth);


                if (obj_intersect)
                {
                    pathSegments[path_index].materialIdHit = objMaterialIdx;

                    scatterRay(pathSegments[path_index].ray,
                               pathSegments[path_index].color,
                               intersect_point,
                               normal,
                               materials[objMaterialIdx],
                               rng,
                               softness);
                }
                else
                {
                    pathSegments[path_index].materialIdHit = geoms[hit_geom_index].materialid;

                    scatterRay(pathSegments[path_index].ray,
                               pathSegments[path_index].color,
                               intersect_point,
                               normal,
                               materials[geoms[hit_geom_index].materialid],
                               rng,
                               softness);
                }


                if (obj_intersect)
                {
                    intersections[path_index].t = t_min;
                    intersections[path_index].materialId = objMaterialIdx; // test material
                    intersections[path_index].surfaceNormal = normal;
                }
                else
                {
                    intersections[path_index].t = t_min;
                    intersections[path_index].materialId = geoms[hit_geom_index].materialid;
                    intersections[path_index].surfaceNormal = normal;
                }
            }
        }
    }
}

__host__ __device__
void traverseKDbare(KDN::NodeBare* nodes, int numNodes,
float& t,
PathSegment pathSegment,
KDN::TriBare* triangles,
glm::vec3& bary,
int& objMaterialIdx,
int& material_size,
glm::vec3& hit,
glm::vec3& norm,
float& t_min,
int& hit_geom_index,
glm::vec3& intersect_point,
glm::vec3& normal,
glm::vec3& tmp_intersect,
glm::vec3& tmp_normal,
bool& obj_intersect,
ShadeableIntersection* intersections,
int* obj_materialOffsets,
int& path_index)
{
    //
    //
    if (numNodes != 0)
    {
        bool nodeIDs[STACK_SIZE] = { false };
        int currID = nodeIDs[nodes[0].ID];
        float dist = -1.0;

        // get the root node
        for (int i = 0; i < numNodes; i++)
        {
            if (nodes[i].parentID == -1)
            {
                currID = nodes[i].ID;
                break;
            }
        }

        KDN::NodeBare* node = &(nodes[currID]);

        bool hitGeom = false;
        float boxdist = -1.0f;
        bary.z = FLT_MAX;
        while (true)
        {
            if (currID == -1)
                break;

            node = &(nodes[currID]);
            // check if it intersects the bounds
            //printf("1\n");

            if (hitGeom == false && node->parentID == -1 && nodeIDs[node->ID] == true)
                break;

            hitGeom = intersectAABBarrays(pathSegment.ray, nodes[currID].mins, nodes[currID].maxs, dist);

            if (nodeIDs[currID] == true)
            {
                nodeIDs[node->ID] = true;
                nodeIDs[node->leftID] = true;
                nodeIDs[node->rightID] = true;
                currID = node->parentID;
                continue;
            }
            else
            {
                //hitGeom = intersectAABB(pathSegment.ray, node->bbox, dist);
                //hitGeom = intersectAABBarrays(pathSegment.ray, nodes[currID].mins, nodes[currID].maxs, dist);

                if (hitGeom == false && node->parentID == -1)
                    break;
            }
            //printf("2\n");

            // if the distance is greater than the last poly hit
            if (hitGeom == false || dist > bary.z)
            {
                nodeIDs[node->ID] = true;
                nodeIDs[node->leftID] = true;
                nodeIDs[node->rightID] = true;
                currID = node->parentID;
            }
            else
            {
                if (nodes[currID].leftID != -1 && nodeIDs[nodes[currID].leftID] != true)
                    currID = node->leftID;
                else if (nodes[currID].rightID != -1 && nodeIDs[nodes[currID].rightID] != true)
                    currID = node->rightID;
                else if (nodeIDs[node->ID] == false)
                {
                    //printf("NODE LOOP: %d\n", nodes[currID].ID);
                    nodeIDs[node->ID] = true;

                    int size = node->triIdSize;
                    if (size > 0)
                    {
                        int start = node->triIdStart;
                        int end = start + size;
                        for (int i = start; i < end; i++)
                        {
                            KDN::TriBare* T = &(triangles[i]);

                            glm::vec3 v1(T->x1, T->y1, T->z1);
                            glm::vec3 v2(T->x2, T->y2, T->z2);
                            glm::vec3 v3(T->x3, T->y3, T->z3);

                            //bary(0.0f, 0.0f, 0.0f);
                            bool intersected = glm::intersectRayTriangle(pathSegment.ray.origin,
                                                                         pathSegment.ray.direction,
                                                                         v1, v2, v3, bary);

                            if (intersected)
                            {
                                glm::vec3 n1(T->nx1, T->ny1, T->nz1);
                                glm::vec3 n2(T->nx2, T->ny2, T->nz2);
                                glm::vec3 n3(T->nx3, T->ny3, T->nz3);

                                objMaterialIdx = triangles[i].mtlIdx + material_size - 1;
                                //(1 - bary.x - bary.y); bary.x; bary.y
                                //printf("material id: %d\n", triangles[i].mtlIdx);
                                hit = pathSegment.ray.origin + pathSegment.ray.direction* bary.z;// (bary2.x * v1 + bary2.y * v2 + bary2.z * v3);
                                norm = glm::normalize((1 - bary.x - bary.y) * n1 + bary.x * n2 + (bary.y) * n3);
                                //norm(glm::normalize(n1));
                                hit += norm*0.0001f;


                                t = glm::distance(pathSegment.ray.origin, hit);

                                if (t > 0.0f && t_min > t)
                                {
                                    t_min = t;
                                    hit_geom_index = obj_materialOffsets[T->mtlIdx];
                                    intersect_point = hit;
                                    normal = norm;
                                    tmp_intersect = hit;
                                    tmp_normal = normal;
                                    obj_intersect = true;
                                    intersections[path_index].t = t;
                                    return;
                                }
                            }
                        }
                    }
                }
                else
                    currID = node->parentID;
            }
        }
    }
}


__host__ __device__
void traverseKDbareShortHybrid(KDN::NodeBare* nodes, int numNodes,
float& t,
PathSegment pathSegment,
KDN::TriBare* triangles,
glm::vec3& bary,
int& objMaterialIdx,
int& material_size,
glm::vec3& hit,
glm::vec3& norm,
float& t_min,
int& hit_geom_index,
glm::vec3& intersect_point,
glm::vec3& normal,
glm::vec3& tmp_intersect,
glm::vec3& tmp_normal,
bool& obj_intersect,
ShadeableIntersection* intersections,
int* obj_materialOffsets,
int& path_index)
{
    //
    //
    if (numNodes != 0)
    {
        bool nodeIDs[STACK_SIZE] = { false };
        int currID = nodeIDs[nodes[0].ID];
        float dist = -1.0;

        // get the root node
        for (int i = 0; i < numNodes; i++)
        {
            if (nodes[i].parentID == -1)
            {
                currID = nodes[i].ID;
                break;
            }
        }

        KDN::NodeBare* node = &(nodes[currID]);

        int axis;
        float tSplit;
        bool hitGeom = false;
        float boxdist = -1.0f;
        bary.z = FLT_MAX;
        while (true)
        {
            if (currID == -1)
                break;

            node = &(nodes[currID]);
            // check if it intersects the bounds
            //printf("1\n");

            if (hitGeom == false && node->parentID == -1 && nodeIDs[node->ID] == true)
                break;

            hitGeom = intersectAABBarrays(pathSegment.ray, nodes[currID].mins, nodes[currID].maxs, dist);

            if (nodeIDs[currID] == true)
            {
                nodeIDs[node->ID] = true;
                nodeIDs[node->leftID] = true;
                nodeIDs[node->rightID] = true;
                currID = node->parentID;
                continue;
            }
            else
            {
                //hitGeom = intersectAABB(pathSegment.ray, node->bbox, dist);
                //hitGeom = intersectAABBarrays(pathSegment.ray, nodes[currID].mins, nodes[currID].maxs, dist);

                if (hitGeom == false && node->parentID == -1)
                    break;
            }
            //printf("2\n");

            // if the distance is greater than the last poly hit
            if (hitGeom == false || dist > bary.z)
            {
                nodeIDs[node->ID] = true;
                nodeIDs[node->leftID] = true;
                nodeIDs[node->rightID] = true;
                currID = node->parentID;
            }
            else
            {
                axis = node->axis;
                //tSplit = (node->splitPos - pathSegment.ray.origin[axis]) / pathSegment.ray.direction[axis];

                if (pathSegment.ray.direction[axis] > 0.0f)// && nodes[node->leftID].splitPos < tSplit)
                {
                    // left side first
                    if (nodes[currID].leftID != -1 && nodeIDs[nodes[currID].leftID] != true)
                        currID = node->leftID;
                    else if (nodes[currID].rightID != -1 && nodeIDs[nodes[currID].rightID] != true)
                        currID = node->rightID;
                    else if (nodeIDs[node->ID] == false)
                    {
                        //printf("NODE LOOP: %d\n", nodes[currID].ID);
                        nodeIDs[node->ID] = true;

                        int size = node->triIdSize;
                        if (size > 0)
                        {
                            int start = node->triIdStart;
                            int end = start + size;
                            for (int i = start; i < end; i++)
                            {
                                KDN::TriBare* T = &(triangles[i]);

                                glm::vec3 v1(T->x1, T->y1, T->z1);
                                glm::vec3 v2(T->x2, T->y2, T->z2);
                                glm::vec3 v3(T->x3, T->y3, T->z3);

                                //bary(0.0f, 0.0f, 0.0f);
                                bool intersected = glm::intersectRayTriangle(pathSegment.ray.origin,
                                                                             pathSegment.ray.direction,
                                                                             v1, v2, v3, bary);

                                if (intersected)
                                {
                                    // skip other side
                                    nodeIDs[nodes[nodeIDs[node->parentID]].rightID] = true;

                                    glm::vec3 n1(T->nx1, T->ny1, T->nz1);
                                    glm::vec3 n2(T->nx2, T->ny2, T->nz2);
                                    glm::vec3 n3(T->nx3, T->ny3, T->nz3);

                                    objMaterialIdx = triangles[i].mtlIdx + material_size - 1;
                                    //(1 - bary.x - bary.y); bary.x; bary.y
                                    //printf("material id: %d\n", triangles[i].mtlIdx);
                                    hit = pathSegment.ray.origin + pathSegment.ray.direction* bary.z;// (bary2.x * v1 + bary2.y * v2 + bary2.z * v3);
                                    norm = glm::normalize((1 - bary.x - bary.y) * n1 + bary.x * n2 + (bary.y) * n3);
                                    //norm(glm::normalize(n1));
                                    hit += norm*0.0001f;


                                    t = glm::distance(pathSegment.ray.origin, hit);

                                    if (t > 0.0f && t_min > t)
                                    {
                                        t_min = t;
                                        hit_geom_index = obj_materialOffsets[T->mtlIdx];
                                        intersect_point = hit;
                                        normal = norm;
                                        tmp_intersect = hit;
                                        tmp_normal = normal;
                                        obj_intersect = true;
                                        intersections[path_index].t = t;
                                        return;
                                    }
                                }
                            }
                        }
                    }
                    else
                        currID = node->parentID;
                }
                else
                {
                    // right side first
                    if (nodes[currID].rightID != -1 && nodeIDs[nodes[currID].rightID] != true)
                        currID = node->rightID;
                    else if (nodes[currID].leftID != -1 && nodeIDs[nodes[currID].leftID] != true)
                        currID = node->leftID;
                    else if (nodeIDs[node->ID] == false)
                    {
                        //printf("NODE LOOP: %d\n", nodes[currID].ID);
                        nodeIDs[node->ID] = true;

                        int size = node->triIdSize;
                        if (size > 0)
                        {
                            int start = node->triIdStart;
                            int end = start + size;
                            for (int i = start; i < end; i++)
                            {
                                KDN::TriBare* T = &(triangles[i]);

                                glm::vec3 v1(T->x1, T->y1, T->z1);
                                glm::vec3 v2(T->x2, T->y2, T->z2);
                                glm::vec3 v3(T->x3, T->y3, T->z3);

                                //bary(0.0f, 0.0f, 0.0f);
                                bool intersected = glm::intersectRayTriangle(pathSegment.ray.origin,
                                                                             pathSegment.ray.direction,
                                                                             v1, v2, v3, bary);

                                if (intersected)
                                {
                                    // skip other side
                                    nodeIDs[nodes[nodeIDs[node->parentID]].leftID] = true;

                                    glm::vec3 n1(T->nx1, T->ny1, T->nz1);
                                    glm::vec3 n2(T->nx2, T->ny2, T->nz2);
                                    glm::vec3 n3(T->nx3, T->ny3, T->nz3);

                                    objMaterialIdx = triangles[i].mtlIdx + material_size - 1;
                                    //(1 - bary.x - bary.y); bary.x; bary.y
                                    //printf("material id: %d\n", triangles[i].mtlIdx);
                                    hit = pathSegment.ray.origin + pathSegment.ray.direction* bary.z;// (bary2.x * v1 + bary2.y * v2 + bary2.z * v3);
                                    norm = glm::normalize((1 - bary.x - bary.y) * n1 + bary.x * n2 + (bary.y) * n3);
                                    //norm(glm::normalize(n1));
                                    hit += norm*0.0001f;


                                    t = glm::distance(pathSegment.ray.origin, hit);

                                    if (t > 0.0f && t_min > t)
                                    {
                                        t_min = t;
                                        hit_geom_index = obj_materialOffsets[T->mtlIdx];
                                        intersect_point = hit;
                                        normal = norm;
                                        tmp_intersect = hit;
                                        tmp_normal = normal;
                                        obj_intersect = true;
                                        intersections[path_index].t = t;
                                        return;
                                    }
                                }
                            }
                        }
                    }
                    else
                        currID = node->parentID;
                }
            }
        }
    }
}


__host__ __device__
void traverseKDshort(KDN::NodeBare* nodes, int numNodes,
float& t,
PathSegment pathSegment,
KDN::TriBare* triangles,
glm::vec3& bary,
int& objMaterialIdx,
int& material_size,
glm::vec3& hit,
glm::vec3& norm,
float& t_min,
int& hit_geom_index,
glm::vec3& intersect_point,
glm::vec3& normal,
glm::vec3& tmp_intersect,
glm::vec3& tmp_normal,
bool& obj_intersect,
ShadeableIntersection* intersections,
int* obj_materialOffsets,
int& path_index)
{
    //printf("numnodes = %d\n", numNodes);


    NodeStack stack[STACK_SIZE];
    int top = -1;



    //std::stack<KDN::NodeBare> nodeStack;
    //KDN::NodeStackBare nodeStack;
    //stack<KDN::NodeBare> nodeStack;
    KDN::NodeBare* node;
    KDN::NodeBare* root;
    KDN::NodeBare* first;
    KDN::NodeBare* second;
    // get the root node
    for (int i = 0; i < numNodes; i++)
    {
        if (nodes[i].parentID == -1)
        {
            node = &(nodes[i]);
            root = &(nodes[i]);
            break;
        }
    }
    //printf("1\n");
    //
    //
    float tMin, tMax, tHit, sceneMax;
    tMin = tMax = 0.0f;
    tHit = t_min;
    sceneMax = FLT_MAX;
    bool pushdown = false;
    int axis = 0;
    float tSplit = 0.0f;

    float dist = 0.0f;
    bool bboxintersect = false;

    while (tMax < sceneMax)
    {
        if (top == -1)
        {
            node = root;
            tMin = tMax;
            tMax = sceneMax;
            pushdown = true;
        }
        else
        {
            node = stack[top].node;
            tMin = node->tmin;
            tMax = node->tmax;
            top--;
            pushdown = false;
        }
        //printf("2\n");
        while (node->triIdSize != 0)
        {
            axis = node->axis;
            tSplit = (node->splitPos - pathSegment.ray.origin[axis]) / pathSegment.ray.direction[axis];

            if (pathSegment.ray.direction[axis] > 0.0f)
            {
                if (nodes[node->leftID].mins[axis] < nodes[node->rightID].mins[axis])
                {
                    first = &(nodes[node->leftID]);
                    second = &(nodes[node->rightID]);
                }
                else
                {
                    first = &nodes[node->rightID];
                    second = &nodes[node->leftID];
                }
            }
            else
            {
                if (nodes[node->leftID].maxs[axis] > nodes[node->rightID].maxs[axis])
                {
                    first = &(nodes[node->leftID]);
                    second = &(nodes[node->rightID]);
                }
                else
                {
                    first = &(nodes[node->rightID]);
                    second = &(nodes[node->leftID]);
                }
            }

            if (tSplit >= tMax || tSplit < 0.0f)
                node = first;
            else if (tSplit <= tMin)
                node = second;
            else
            {
                second->tmin = tSplit;
                second->tmax = tMax;
                top++;
                if (top <= 199)
                {
                    stack[top].node = second;
                    stack[top].tmin = tSplit;
                    stack[top].tmax = tMax;
                }
                else
                {
                    //printf("bogus");
                    break;

                }
                node = first;
                tMax = tSplit;
                pushdown = false;
            }
            if (pushdown)
                root = node;


            bboxintersect = intersectAABBarrays(pathSegment.ray, node->mins, node->maxs, dist);
            if (bboxintersect)
            {
                int start = node->triIdStart;
                int end = start + node->triIdSize;
                for (int i = start; i < end; i++)
                {
                    KDN::TriBare* T = &(triangles[i]);

                    glm::vec3 v1(T->x1, T->y1, T->z1);
                    glm::vec3 v2(T->x2, T->y2, T->z2);
                    glm::vec3 v3(T->x3, T->y3, T->z3);

                    //bary(0.0f, 0.0f, 0.0f);
                    bool intersected = glm::intersectRayTriangle(pathSegment.ray.origin,
                                                                 pathSegment.ray.direction,
                                                                 v1, v2, v3, bary);

                    if (intersected)
                    {
                        glm::vec3 n1(T->nx1, T->ny1, T->nz1);
                        glm::vec3 n2(T->nx2, T->ny2, T->nz2);
                        glm::vec3 n3(T->nx3, T->ny3, T->nz3);

                        //printf("5\n");
                        objMaterialIdx = triangles[i].mtlIdx + material_size - 1;
                        //(1 - bary.x - bary.y); bary.x; bary.y
                        //printf("material id: %d\n", triangles[i].mtlIdx);
                        hit = pathSegment.ray.origin + pathSegment.ray.direction* bary.z;// (bary2.x * v1 + bary2.y * v2 + bary2.z * v3);
                        norm = -glm::normalize((1 - bary.x - bary.y) * n1 + bary.x * n2 + (bary.y) * n3);
                        //norm(glm::normalize(n1));
                        hit += norm*0.0001f;


                        t = glm::distance(pathSegment.ray.origin, hit);

                        if (t > 0.0f || t_min > t)
                        {
                            tHit = min(tHit, t);

                            t_min = t;
                            hit_geom_index = obj_materialOffsets[T->mtlIdx];
                            intersect_point = hit;
                            normal = norm;
                            tmp_intersect = hit;
                            tmp_normal = normal;
                            obj_intersect = true;
                            intersections[path_index].t = t;
                            return;
                        }
                    }
                }
            }
        }
    }
}







__host__ __device__
void traverseKD(KDN::NodeBare* nodes, int numNodes,
float& t,
PathSegment pathSegment,
KDN::TriBare* triangles,
glm::vec3& bary,
int& objMaterialIdx,
int& material_size,
glm::vec3& hit,
glm::vec3& norm,
float& t_min,
int& hit_geom_index,
glm::vec3& intersect_point,
glm::vec3& normal,
glm::vec3& tmp_intersect,
glm::vec3& tmp_normal,
bool& obj_intersect,
ShadeableIntersection* intersections,
int* obj_materialOffsets,
int& path_index)
{
    //printf("numnodes = %d\n", numNodes);


    NodeStack stack[STACK_SIZE];
    int top = -1;



    //std::stack<KDN::NodeBare> nodeStack;
    //KDN::NodeStackBare nodeStack;
    //stack<KDN::NodeBare> nodeStack;
    KDN::NodeBare* node;
    KDN::NodeBare* root;
    KDN::NodeBare* first;
    KDN::NodeBare* second;
    // get the root node
    for (int i = 0; i < numNodes; i++)
    {
        if (nodes[i].parentID == -1)
        {
            node = &(nodes[i]);
            root = &(nodes[i]);
            break;
        }
    }
    //printf("1\n");
    //
    //
    glm::vec3 origin = pathSegment.ray.origin;
    glm::vec3 invDirection(1.0f / pathSegment.ray.direction[0],
                           1.0f / pathSegment.ray.direction[1],
                           1.0f / pathSegment.ray.direction[2]);

    float tmax = FLT_MAX;
    float tClosestIntersection = t_min;
    bool notFullyTraversed = true;


    while (notFullyTraversed)
    {
        if (node->triIdSize != 0)
        {
            //test all primitives inside the leaf
            float dist = 0.0f;
            bool bboxintersect = intersectAABBarrays(pathSegment.ray, node->mins, node->maxs, dist);
            if (bboxintersect)
            {
                int start = node->triIdStart;
                int end = start + node->triIdSize;
                for (int i = start; i < end; i++)
                {
                    KDN::TriBare* T = &(triangles[i]);

                    glm::vec3 v1(T->x1, T->y1, T->z1);
                    glm::vec3 v2(T->x2, T->y2, T->z2);
                    glm::vec3 v3(T->x3, T->y3, T->z3);

                    //bary(0.0f, 0.0f, 0.0f);
                    bool intersected = glm::intersectRayTriangle(pathSegment.ray.origin,
                                                                 pathSegment.ray.direction,
                                                                 v1, v2, v3, bary);

                    if (intersected)
                    {
                        glm::vec3 n1(T->nx1, T->ny1, T->nz1);
                        glm::vec3 n2(T->nx2, T->ny2, T->nz2);
                        glm::vec3 n3(T->nx3, T->ny3, T->nz3);

                        //printf("5\n");
                        objMaterialIdx = triangles[i].mtlIdx + material_size - 1;
                        //(1 - bary.x - bary.y); bary.x; bary.y
                        //printf("material id: %d\n", triangles[i].mtlIdx);
                        hit = pathSegment.ray.origin + pathSegment.ray.direction* bary.z;// (bary2.x * v1 + bary2.y * v2 + bary2.z * v3);
                        norm = -glm::normalize((1 - bary.x - bary.y) * n1 + bary.x * n2 + (bary.y) * n3);
                        //norm(glm::normalize(n1));
                        hit += norm*0.0001f;


                        t = glm::distance(pathSegment.ray.origin, hit);

                        if (t > 0.0f && t_min > t)
                        {
                            //tHit = min(tHit, t);

                            t_min = t;
                            hit_geom_index = obj_materialOffsets[T->mtlIdx];
                            intersect_point = hit;
                            normal = norm;
                            tmp_intersect = hit;
                            tmp_normal = normal;
                            obj_intersect = true;
                            intersections[path_index].t = t;
                            return;
                        }
                    }
                }
            }

            //test if leaf + empty stack => return
            if (top == -1)
            {
                notFullyTraversed = false;
            }
            else
            {
                //pop all stack
                origin = stack[top].origin;
                tmax = stack[top].tmax;
                node = stack[top].node;
                top--;
            }
        }
        else
        {
            //get axis of node and its split plane
            const int axis = node->axis;
            const float plane = node->splitPos;

            //test if ray is not parallel to plane
            if ((fabs(pathSegment.ray.direction[axis]) > EPSILON))
            {
                const float t = (plane - origin[axis]) * invDirection[axis];

                //case of the ray intersecting the plane, then test both childs
                if (0.0f < t && t < tmax) {
                    //traverse near first, then far. Set tmax = t for near

                    //push only far child onto stack
                    top++;
                    stack[top].origin[0] = origin[0] + pathSegment.ray.direction[0] * t;
                    stack[top].origin[1] = origin[1] + pathSegment.ray.direction[1] * t;
                    stack[top].origin[2] = origin[2] + pathSegment.ray.direction[2] * t;
                    stack[top].node = (origin[axis] > plane) ? &(nodes[node->leftID]) : &(nodes[node->rightID]);
                    stack[top].tmax = tmax - t;

                    tmax = t;
                }
            }
            //in every case: traverse near child first
            node = (origin[axis] > plane) ? &(nodes[node->rightID]) : &(nodes[node->leftID]);

        }
    }
}


// pathTraceOneBounce handles ray intersections, generate intersections for shading, 
// and scatter new ray. You might want to call scatterRay from interactions.h
__global__ void pathTraceOneBounceKDbare(
    int depth
    , int iter
    , int num_paths
    , PathSegment * pathSegments
    , Geom * geoms
    , int geoms_size
    , Material * materials
    , int material_size
    , ShadeableIntersection * intersections
    , float softness
    , KDN::TriBare* triangles
    , int numTriangles
    , KDN::NodeBare* nodes
    , int numNodes
    , int* obj_materialOffsets
    , int hasobj
    )
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        //path_index = pathSegments[path_index].pixelIndex;
        PathSegment pathSegment = pathSegments[path_index];
        //printf("\nO1");
        if (pathSegments[path_index].remainingBounces>0)
        {
            float t;
            glm::vec3 intersect_point;
            glm::vec3 normal;
            float t_min = FLT_MAX;
            int hit_geom_index = -1;
            bool outside = true;

            glm::vec3 tmp_intersect;
            glm::vec3 tmp_normal;

            glm::vec3 hit;
            glm::vec3 norm;
            glm::vec3 bary;
            glm::vec3 v1;
            glm::vec3 v2;
            glm::vec3 v3;
            glm::vec3 n1;
            glm::vec3 n2;
            glm::vec3 n3;
            int pidxo1 = 0;
            int pidxo2 = 0;
            int pidxo3 = 0;
            bool intersected = false;
            bool obj_intersect = false;
            // naive parse through global geoms
            //printf("\nO2");

            int objMaterialIdx = -1;
            for (int i = 0; i < geoms_size; i++)
            {
                Geom & geom = geoms[i];

                if (geom.type == CUBE)
                {
                    t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }
                else if (geom.type == SPHERE)
                {
                    t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }

                // Compute the minimum t from the intersection tests to determine what
                // scene geometry object was hit first.
                if (t > 0.0f && t_min > t)
                {
                    t_min = t;
                    hit_geom_index = i;
                    intersect_point = tmp_intersect;
                    normal = tmp_normal;
                }
            }

            objMaterialIdx = -1;
            int iterator = 0;
            if (hasobj)
            {
                /*
                traverseKDbare(nodes, numNodes, t,
                                pathSegment, triangles,
                                bary, objMaterialIdx,
                                material_size, hit,
                                norm, t_min,
                                hit_geom_index, intersect_point,
                                normal, tmp_intersect,
                                tmp_normal, obj_intersect,
                                intersections, obj_materialOffsets,
                                path_index);
                */
                ///*
                traverseKDbareShortHybrid(nodes, numNodes, t,
                                pathSegment, triangles,
                                bary, objMaterialIdx,
                                material_size, hit,
                                norm, t_min,
                                hit_geom_index, intersect_point,
                                normal, tmp_intersect,
                                tmp_normal, obj_intersect,
                                intersections, obj_materialOffsets,
                                path_index);
                //*/
            }

            if (hit_geom_index == -1)
            {
                intersections[path_index].t = -1.0f;
            }
            else
            {

                // updating rays
                //thrust::default_random_engine rng = makeSeededRandomEngine(iter, depth, depth); // WAY TOO COOL!
                thrust::default_random_engine rng = makeSeededRandomEngine(iter, path_index, depth);


                if (obj_intersect)
                {
                    pathSegments[path_index].materialIdHit = objMaterialIdx;

                    scatterRay(pathSegments[path_index].ray,
                               pathSegments[path_index].color,
                               intersect_point,
                               normal,
                               materials[objMaterialIdx],
                               rng,
                               softness);
                }
                else
                {
                    pathSegments[path_index].materialIdHit = geoms[hit_geom_index].materialid;

                    scatterRay(pathSegments[path_index].ray,
                               pathSegments[path_index].color,
                               intersect_point,
                               normal,
                               materials[geoms[hit_geom_index].materialid],
                               rng,
                               softness);
                }


                if (obj_intersect)
                {
                    intersections[path_index].t = t_min;
                    intersections[path_index].materialId = objMaterialIdx; // test material
                    intersections[path_index].surfaceNormal = normal;
                }
                else
                {
                    intersections[path_index].t = t_min;
                    intersections[path_index].materialId = geoms[hit_geom_index].materialid;
                    intersections[path_index].surfaceNormal = normal;
                }
            }
        }
    }
}

// pathTraceOneBounce handles ray intersections, generate intersections for shading, 
// and scatter new ray. You might want to call scatterRay from interactions.h
__global__ void pathTraceOneBounceKDbareBoxes(
    int depth
    , int iter
    , int num_paths
    , PathSegment * pathSegments
    , Geom * geoms
    , int geoms_size
    , Material * materials
    , int material_size
    , ShadeableIntersection * intersections
    , float softness
    , KDN::TriBare* triangles
    , int numTriangles
    , KDN::NodeBare* nodes
    , int numNodes
    , int* obj_materialOffsets
    , int hasobj
    )
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        //path_index = pathSegments[path_index].pixelIndex;
        PathSegment pathSegment = pathSegments[path_index];
        //printf("\nO1");
        if (pathSegments[path_index].remainingBounces>0)
        {
            float t;
            glm::vec3 intersect_point;
            glm::vec3 normal;
            float t_min = FLT_MAX;
            int hit_geom_index = -1;
            bool outside = true;

            glm::vec3 tmp_intersect;
            glm::vec3 tmp_normal;

            glm::vec3 hit;
            glm::vec3 norm;
            glm::vec3 bary;
            glm::vec3 v1;
            glm::vec3 v2;
            glm::vec3 v3;
            glm::vec3 n1;
            glm::vec3 n2;
            glm::vec3 n3;
            int pidxo1 = 0;
            int pidxo2 = 0;
            int pidxo3 = 0;
            bool intersected = false;
            bool obj_intersect = false;
            // naive parse through global geoms
            //printf("\nO2");

            int objMaterialIdx = -1;
            for (int i = 0; i < geoms_size; i++)
            {
                Geom & geom = geoms[i];

                if (geom.type == CUBE)
                {
                    t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }
                else if (geom.type == SPHERE)
                {
                    t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }

                // Compute the minimum t from the intersection tests to determine what
                // scene geometry object was hit first.
                if (t > 0.0f && t_min > t)
                {
                    t_min = t;
                    hit_geom_index = i;
                    intersect_point = tmp_intersect;
                    normal = tmp_normal;
                }
            }

            objMaterialIdx = -1;
            int iterator = 0;
            if (hasobj)
            {
                /*
                traverseKDbare(nodes, numNodes, t,
                pathSegment, triangles,
                bary, objMaterialIdx,
                material_size, hit,
                norm, t_min,
                hit_geom_index, intersect_point,
                normal, tmp_intersect,
                tmp_normal, obj_intersect,
                intersections, obj_materialOffsets,
                path_index);
                */

                for (int i = 0; i < numNodes; i++)
                {
                    t = boxIntersectionTestBox(nodes[i].mins, nodes[i].maxs, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                    // Compute the minimum t from the intersection tests to determine what
                    // scene geometry object was hit first.
                    if (t > 0.0f && t_min > t)
                    {
                        t_min = t;
                        hit_geom_index = geoms_size;
                        intersect_point = tmp_intersect;
                        normal = tmp_normal;
                        obj_intersect = true;
                        objMaterialIdx = material_size - 1;
                    }
                }
            }

            if (hit_geom_index == -1)
            {
                intersections[path_index].t = -1.0f;
            }
            else
            {

                // updating rays
                //thrust::default_random_engine rng = makeSeededRandomEngine(iter, depth, depth); // WAY TOO COOL!
                thrust::default_random_engine rng = makeSeededRandomEngine(iter, path_index, depth);


                if (obj_intersect)
                {
                    pathSegments[path_index].materialIdHit = objMaterialIdx;

                    scatterRay(pathSegments[path_index].ray,
                               pathSegments[path_index].color,
                               intersect_point,
                               normal,
                               materials[objMaterialIdx],
                               rng,
                               softness);
                }
                else
                {
                    pathSegments[path_index].materialIdHit = geoms[hit_geom_index].materialid;

                    scatterRay(pathSegments[path_index].ray,
                               pathSegments[path_index].color,
                               intersect_point,
                               normal,
                               materials[geoms[hit_geom_index].materialid],
                               rng,
                               softness);
                }


                if (obj_intersect)
                {
                    intersections[path_index].t = t_min;
                    intersections[path_index].materialId = objMaterialIdx; // test material
                    intersections[path_index].surfaceNormal = normal;
                }
                else
                {
                    intersections[path_index].t = t_min;
                    intersections[path_index].materialId = geoms[hit_geom_index].materialid;
                    intersections[path_index].surfaceNormal = normal;
                }
            }
        }
    }
}






// pathTraceOneBounce handles ray intersections, generate intersections for shading, 
// and scatter new ray. You might want to call scatterRay from interactions.h
__global__ void pathTraceOneBounceKDbareShortStack(
    int depth
    , int iter
    , int num_paths
    , PathSegment * pathSegments
    , Geom * geoms
    , int geoms_size
    , Material * materials
    , int material_size
    , ShadeableIntersection * intersections
    , float softness
    , KDN::TriBare* triangles
    , int numTriangles
    , KDN::NodeBare* nodes
    , int numNodes
    , int* obj_materialOffsets
    , int hasobj
    )
{

    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        //path_index = pathSegments[path_index].pixelIndex;
        PathSegment pathSegment = pathSegments[path_index];
        //printf("\nO1");
        if (pathSegments[path_index].remainingBounces>0)
        {
            float t = 0.0;
            glm::vec3 intersect_point;
            glm::vec3 normal;
            float t_min = FLT_MAX;
            int hit_geom_index = -1;
            bool outside = true;

            glm::vec3 tmp_intersect;
            glm::vec3 tmp_normal;

            glm::vec3 hit;
            glm::vec3 norm;
            glm::vec3 bary;
            glm::vec3 v1;
            glm::vec3 v2;
            glm::vec3 v3;
            glm::vec3 n1;
            glm::vec3 n2;
            glm::vec3 n3;
            int pidxo1 = 0;
            int pidxo2 = 0;
            int pidxo3 = 0;
            bool intersected = false;
            bool obj_intersect = false;
            // naive parse through global geoms
            //printf("\nO2");

            int objMaterialIdx = -1;
            for (int i = 0; i < geoms_size; i++)
            {
                Geom & geom = geoms[i];

                if (geom.type == CUBE)
                {
                    t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }
                else if (geom.type == SPHERE)
                {
                    t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }

                // Compute the minimum t from the intersection tests to determine what
                // scene geometry object was hit first.
                if (t > 0.0f && t_min > t)
                {
                    t_min = t;
                    hit_geom_index = i;
                    intersect_point = tmp_intersect;
                    normal = tmp_normal;
                }
            }

            objMaterialIdx = -1;
            int iterator = 0;
            if (hasobj)
            {
                //printf("before traverse\n");
                ///*
                traverseKDshort(nodes, numNodes, t,
                           pathSegment, triangles,
                           bary, objMaterialIdx,
                           material_size, hit,
                           norm, t_min,
                           hit_geom_index, intersect_point,
                           normal, tmp_intersect,
                           tmp_normal, obj_intersect,
                           intersections, obj_materialOffsets,
                           path_index);
                //*/
                /*
                traverseKD(nodes, numNodes, t,
                                pathSegment, triangles,
                                bary, objMaterialIdx,
                                material_size, hit,
                                norm, t_min,
                                hit_geom_index, intersect_point,
                                normal, tmp_intersect,
                                tmp_normal, obj_intersect,
                                intersections, obj_materialOffsets,
                                path_index);
                */
                
                //printf("after traverse\n");
                
                /*
                thrust::device_vector<KDN::NodeBare> nodeStack;
                //KDN::NodeStackBare nodeStack;
                //stack<KDN::NodeBare> nodeStack;
                KDN::NodeBare node;
                KDN::NodeBare root;
                KDN::NodeBare first;
                KDN::NodeBare second;
                // get the root node
                for (int i = 0; i < numNodes; i++)
                {
                    if (nodes[i].parentID == -1)
                    {
                        node = nodes[i];
                        root = nodes[i];
                        break;
                    }
                }
                //
                //
                float tMin, tMax, tHit;
                tMin = tMax = 0.0f;
                tHit = t;
                bool pushdown = false;
                int axis = 0;
                float tSplit = 0.0f;

                while (tMax < t)
                {
                    if (nodeStack.empty())
                    {
                        node = root;
                        tMin = tMax;
                        tMax = t;
                        pushdown = true;
                    }
                    else
                    {
                        node = nodeStack.end()[0];
                        tMin = node.tmin;
                        tMax = node.tmax;
                        nodeStack.pop_back();
                        pushdown = false;
                    }
                    while (node.triIdSize != 0)
                    {
                        axis = node.axis;
                        tSplit = (node.splitPos - pathSegment.ray.origin[axis]) / pathSegment.ray.direction[axis];

                        if (pathSegment.ray.direction[axis] > 0)
                        {
                            if (nodes[node.leftID].mins[axis] < nodes[node.rightID].mins[axis])
                            {
                                first = nodes[node.leftID];
                                second = nodes[node.rightID];
                            }
                            else
                            {
                                first = nodes[node.rightID];
                                second = nodes[node.leftID];
                            }
                        }
                        else
                        {
                            if (nodes[node.leftID].mins[axis] > nodes[node.rightID].mins[axis])
                            {
                                first = nodes[node.leftID];
                                second = nodes[node.rightID];
                            }
                            else
                            {
                                first = nodes[node.rightID];
                                second = nodes[node.leftID];
                            }
                        }

                        if (tSplit >= tMax || tSplit < 0)
                            node = first;
                        else if (tSplit <= tMin)
                            node = second;
                        else
                        {
                            second.tmin = tMin;
                            second.tmax = tMax;
                            nodeStack.push_back(second);
                            node = first;
                            tMax = tSplit;
                            pushdown = false;
                        }
                        if (pushdown)
                            root = node;



                        int start = node.triIdStart;
                        int end = start + node.triIdSize;
                        for (int i = start; i < end; i++)
                        {
                            KDN::TriBare* T = &(triangles[i]);

                            glm::vec3 v1(T->x1, T->y1, T->z1);
                            glm::vec3 v2(T->x2, T->y2, T->z2);
                            glm::vec3 v3(T->x3, T->y3, T->z3);

                            glm::vec3 n1(T->nx1, T->ny1, T->nz1);
                            glm::vec3 n2(T->nx2, T->ny2, T->nz2);
                            glm::vec3 n3(T->nx3, T->ny3, T->nz3);

                            //bary(0.0f, 0.0f, 0.0f);
                            bool intersected = glm::intersectRayTriangle(pathSegment.ray.origin,
                                                                            pathSegment.ray.direction,
                                                                            v1, v2, v3, bary);

                            if (intersected)
                            {
                                objMaterialIdx = triangles[i].mtlIdx + material_size - 1;
                                //(1 - bary.x - bary.y); bary.x; bary.y
                                //printf("material id: %d\n", triangles[i].mtlIdx);
                                hit = pathSegment.ray.origin + pathSegment.ray.direction* bary.z;// (bary2.x * v1 + bary2.y * v2 + bary2.z * v3);
                                norm = -glm::normalize((1 - bary.x - bary.y) * n1 + bary.x * n2 + (bary.y) * n3);
                                //norm(glm::normalize(n1));
                                hit += norm*0.0001f;


                                t = glm::distance(pathSegment.ray.origin, hit);

                                if (t > 0.0f && t_min > t)
                                {
                                    tHit = min(tHit, t);

                                    t_min = t;
                                    hit_geom_index = obj_materialOffsets[T->mtlIdx];
                                    intersect_point = hit;
                                    normal = norm;
                                    tmp_intersect = hit;
                                    tmp_normal = normal;
                                    obj_intersect = true;
                                    intersections[path_index].t = t;
                                }
                            }
                        }
                    }
                }*/
            }



            if (hit_geom_index == -1)
            {
                intersections[path_index].t = -1.0f;
            }
            else
            {

                // updating rays
                //thrust::default_random_engine rng = makeSeededRandomEngine(iter, depth, depth); // WAY TOO COOL!
                thrust::default_random_engine rng = makeSeededRandomEngine(iter, path_index, depth);


                if (obj_intersect)
                {
                    pathSegments[path_index].materialIdHit = objMaterialIdx;

                    scatterRay(pathSegments[path_index].ray,
                               pathSegments[path_index].color,
                               intersect_point,
                               normal,
                               materials[objMaterialIdx],
                               rng,
                               softness);
                }
                else
                {
                    pathSegments[path_index].materialIdHit = geoms[hit_geom_index].materialid;

                    scatterRay(pathSegments[path_index].ray,
                               pathSegments[path_index].color,
                               intersect_point,
                               normal,
                               materials[geoms[hit_geom_index].materialid],
                               rng,
                               softness);
                }


                if (obj_intersect)
                {
                    intersections[path_index].t = t_min;
                    intersections[path_index].materialId = objMaterialIdx; // test material
                    intersections[path_index].surfaceNormal = normal;
                }
                else
                {
                    intersections[path_index].t = t_min;
                    intersections[path_index].materialId = geoms[hit_geom_index].materialid;
                    intersections[path_index].surfaceNormal = normal;
                }
            }
        }
    }
}

// pathTraceOneBounce handles ray intersections, generate intersections for shading, 
// This is the KD-tree implementation
__global__ void pathTraceOneBounceKD(
    int depth
    , int iter
    , int num_paths
    , PathSegment * pathSegments
    , Geom * geoms
    , int geoms_size
    , Material * materials
    , int material_size
    , ShadeableIntersection * intersections
    , float softness
    , KDN::Triangle* triangles
    , int numTriangles
    , KDN::KDnode* nodes
    , int numNodes
    , int hasobj
    )
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    //printf("tracekd 1\n");
    if (path_index < num_paths)
    {
        //path_index = pathSegments[path_index].pixelIndex;
        PathSegment pathSegment = pathSegments[path_index];
        //printf("\nO1");
        if (pathSegments[path_index].remainingBounces>0)
        {
            //printf("tracekd 2\n");
            float t;
            glm::vec3 intersect_point;
            glm::vec3 normal;
            float t_min = FLT_MAX;
            int hit_geom_index = -1;
            bool outside = true;

            glm::vec3 tmp_intersect;
            glm::vec3 tmp_normal;

            glm::vec3 hit;
            glm::vec3 norm;
            glm::vec3 bary;
            glm::vec3 v1;
            glm::vec3 v2;
            glm::vec3 v3;
            glm::vec3 n1;
            glm::vec3 n2;
            glm::vec3 n3;
            int pidxo1 = 0;
            int pidxo2 = 0;
            int pidxo3 = 0;
            bool intersected = false;
            bool obj_intersect = false;
            // naive parse through global geoms
            //printf("\nO2");

            //printf("tracekd 3\n");
            int objMaterialIdx = -1;
            for (int i = 0; i < geoms_size; i++)
            {
                Geom & geom = geoms[i];

                if (geom.type == CUBE)
                {
                    t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }
                else if (geom.type == SPHERE)
                {
                    t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                }

                // TODO KDTREE TRAVERSAL

                // Compute the minimum t from the intersection tests to determine what
                // scene geometry object was hit first.
                
                if (t > 0.0f && t_min > t)
                {
                    t_min = t;
                    hit_geom_index = i;
                    intersect_point = tmp_intersect;
                    normal = tmp_normal;
                }
                
            }

            //printf("tracekd 4\n");

            // start polygon hits
            //t_min = FLT_MAX;
            //for (int i = 0; i < obj_numshapes; i++)
            //    printf("\noffset = %d", obj_polyoffsets[i]);


            //printf("\nO3");
            //printf("\nNUMSHAPES = %d\n", obj_numshapes);
            objMaterialIdx = -1;
            int iterator = 0;
            if (hasobj)
            {
                //printf("tracekd 5\n");
                float dist = -1.0f;
                glm::vec3 norm;

                //printf("BEFORE ALLOCATING\n");
                //bool* nodeIDs = new bool[numNodes];
                // delete [] nodeIDs;

                
                //printf("AFTER ALLOCATING\n");

                //printf("BEFORE INTERSECT\n");
                //glm::vec4 intersection = intersectKDLoopDeref(pathSegment.ray, nodes, numNodes, triangles, numTriangles);
                //norm.x = intersection.x;
                //norm.y = intersection.y;
                //norm.z = intersection.z;
                //dist = intersection.w;










                dist = -1.0f;
                norm = glm::vec3(0.0f);
                bool hitGeom = false;

                Ray r = pathSegment.ray;

                // USE AN ARRAY OF 0 NODE IDS AND SET THEM TO 1 once they're visited
                // instead of using visited to avoid conflicts when reading from
                // multiple threads
                bool nodeIDs[1000] = { false };
                //memset(nodeIDs, 0, sizeof(bool)*numNodes);



                if (numNodes != 0)
                {
                    float mindist = FLT_MAX;
                    int currID = nodeIDs[nodes[0].ID];

                    // get the root node
                    for (int i = 0; i < numNodes; i++)
                    {
                        if (nodes[i].parentID == -1)
                        {
                            currID = nodes[i].ID;
                            break;
                        }
                    }

                    float boxdist = -1.0f;
                    while (true)
                    {
                        if (currID == -1)
                            break;

                        // check if it intersects the bounds
                        //printf("1\n");
                        hitGeom = intersectAABB(r, nodes[currID].bbox, dist);
                        //printf("2\n");

                        if (hitGeom == false)
                        {
                            nodeIDs[nodes[currID].ID] = true;
                            currID = nodes[currID].parentID;
                        }
                        else
                        {
                            if (nodes[currID].leftID != -1 && nodeIDs[nodes[currID].leftID] != true)
                                currID = nodes[currID].leftID;
                            else if (nodes[currID].rightID != -1 && nodeIDs[nodes[currID].rightID] != true)
                                currID = nodes[currID].rightID;
                            else if (nodeIDs[nodes[currID].ID] == false)
                            {
                                //std::cout << "NODE LOOP: " << nodes[currID].ID << " PARENT: " << nodes[currID].parentID << std::endl;
                                nodeIDs[nodes[currID].ID] = true;

                                int size = nodes[currID].triIdSize;
                                if (size > 0)
                                {
                                    int start = nodes[currID].triIdStart;
                                    int end = start + size;
                                    for (int i = start; i < end; i++)
                                    {
                                        //KDN::Triangle t = triangles[i];

                                        glm::vec3 v1(triangles[i].x1, triangles[i].y1, triangles[i].z1);
                                        glm::vec3 v2(triangles[i].x2, triangles[i].y2, triangles[i].z2);
                                        glm::vec3 v3(triangles[i].x3, triangles[i].y3, triangles[i].z3);

                                        glm::vec3 barytemp(0.0f, 0.0f, 0.0f);
                                        bool intersected = glm::intersectRayTriangle(r.origin,
                                                                                     r.direction,
                                                                                     v1, v2, v3, barytemp);

                                        if (intersected && barytemp.z < mindist)
                                        {
                                            glm::vec3 bary(barytemp.x, barytemp.y, 1.0 - barytemp.x - barytemp.y);

                                            glm::vec3 n1(triangles[i].nx1, triangles[i].ny1, triangles[i].nz1);
                                            glm::vec3 n2(triangles[i].nx2, triangles[i].ny2, triangles[i].nz2);
                                            glm::vec3 n3(triangles[i].nx3, triangles[i].ny3, triangles[i].nz3);
                                            norm = (bary[0] * n1 + bary[1] * n2 + bary[2] * n3);

                                            dist = barytemp.z;
                                            mindist = dist;
                                            //glm::vec3 pos = r.origin + r.direction * dist;

                                            glm::vec3 intersect = r.origin + r.direction*dist;
                                            //printf("KDLOOPPTR INTERSECT POINT: P: [%f %f %f] NODEID: %d\n", intersect.x,
                                            //       intersect.y,
                                            //       intersect.z,
                                            //       currID);


                                            norm = glm::normalize((1 - bary.x - bary.y) * n1 + bary.x * n2 + (bary.y) * n3);
                                            //norm(glm::normalize(n1));
                                            //intersect += norm*0.0001f;

                                                
                                            t = dist;
                                                
                                            if (t > 0.0f && t_min > t)
                                            {
                                                t_min = t;
                                                hit_geom_index = 0;// obj_materialOffsets[i];
                                                intersect_point = intersect;
                                                tmp_intersect = intersect;
                                                tmp_normal = norm;//glm::vec3(0.0f, 1.0f, 0.0f);
                                                //obj_intersect = true;
                                                intersections[path_index].t = t;
                                            }
                                        }
                                    }
                                }
                            }
                            else
                                currID = nodes[currID].parentID;
                        }
                    }
                }

                //printf("AFTER INTERSECT\n");
                //printf("tracekd 5.1\n");
                if (hit_geom_index != -1)
                {
                    hit_geom_index = 0;
                    obj_intersect = true;
                    t_min = dist;
                    intersect_point = tmp_intersect;
                    //printf("tracekd 5.2\n");
                    // TODO add normals to Triangle class and get them
                    // testing with default normal
                    normal = tmp_normal;
                    //printf("tracekd 5.3\n");
                }
            }

            //printf("tracekd 6\n");

            //printf("\nO11");

            // TODO: scatter the ray, generate intersections for shading
            // feel free to modify the code below

            if (hit_geom_index == -1)
            {
                intersections[path_index].t = -1.0f;
            }
            else
            {
                //The ray hits something
                //intersections[path_index].t = t_min;
                //intersections[path_index].materialId = geoms[hit_geom_index].materialid;
                //intersections[path_index].surfaceNormal = normal;


                // updating rays
                //thrust::default_random_engine rng = makeSeededRandomEngine(iter, depth, depth); // WAY TOO COOL!
                thrust::default_random_engine rng = makeSeededRandomEngine(iter, path_index, depth);

                //printf("tracekd 7\n");
                if (obj_intersect)// && false)
                {
                    // testing for now;
                    objMaterialIdx = 0;

                    pathSegments[path_index].materialIdHit = objMaterialIdx;

                    scatterRay(pathSegments[path_index].ray,
                               pathSegments[path_index].color,
                               intersect_point,
                               normal,
                               materials[objMaterialIdx],
                               rng,
                               softness);
                }
                else
                {
                    pathSegments[path_index].materialIdHit = geoms[hit_geom_index].materialid;

                    scatterRay(pathSegments[path_index].ray,
                               pathSegments[path_index].color,
                               intersect_point,
                               normal,
                               materials[geoms[hit_geom_index].materialid],
                               rng,
                               softness);
                }
                //printf("tracekd 8\n");
                if (obj_intersect)// && false)
                {
                    intersections[path_index].t = t_min;
                    intersections[path_index].materialId = geoms[0].materialid; //objMaterialIdx; // test material
                    intersections[path_index].surfaceNormal = normal;
                }
                else
                {
                    intersections[path_index].t = t_min;
                    intersections[path_index].materialId = geoms[hit_geom_index].materialid;
                    intersections[path_index].surfaceNormal = normal;
                }
                //printf("tracekd 9\n");
            }
        }
    }
}


__global__ void shadeMaterial(
    int iter
    , int num_paths
    , ShadeableIntersection * shadeableIntersections
    , PathSegment * pathSegments
    , Material * materials
    , bool enablesss
    )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        //idx = pathSegments[idx].initialidx;
        //idx = pathSegments[idx].pixelIndex;
        if (pathSegments[idx].remainingBounces>0)
        {
            ShadeableIntersection intersection = shadeableIntersections[idx];
            if (intersection.t > 0.0f) 
            { // if the intersection exists...
                // Set up the RNG
                thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
                thrust::uniform_real_distribution<float> u01(0, 1);

                Material material = materials[intersection.materialId];
                glm::vec3 materialColor = material.color;

                // If the material indicates that the object was a light, "light" the ray
                if (material.emittance > 0.0f) {
                    pathSegments[idx].color *= (materialColor * material.emittance);
                    pathSegments[idx].remainingBounces = 0;
                }
                // Otherwise, do some pseudo-lighting computation. This is actually more
                // like what you would expect from shading in a rasterizer like OpenGL.
                else {
                    //float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));

                    //if (pathSegments[idx].ray.isrefr)
                    //{
                    //    pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + materialColor * 0.7f + material.hasRefractive * materialColor;
                    //}

                    //else if (pathSegments[idx].ray.isrefl)
                    //{
                    //    pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + materialColor * 0.7f + material.hasReflective * materialColor;
                    //}

                    if (enablesss && (material.transmittance.x > 0.0f || material.transmittance.y > 0.0f || material.transmittance.z > 0.0f))
                    {
                        float scenescale = 1.0f;
                        float sss = scenescale * pathSegments[idx].ray.sdepth > 1.0 ? 1.0 : pathSegments[idx].ray.sdepth;
                        sss = 1.0f - sss < 0.0 ? 0.0 : sss;
                        sss = glm::pow(sss, 2);
                        pathSegments[idx].color *= (materialColor)* 1.0f + material.hasRefractive * material.specular.color + sss * material.transmittance;
                    }
                    else if (material.hasRefractive > 0.0f)
                    {
                        pathSegments[idx].color *= (materialColor)* 1.0f + material.hasRefractive * material.specular.color;
                    }
                    else if (material.hasReflective > 0.0f)
                    {
                        pathSegments[idx].color *= (materialColor)* 1.0f + material.hasReflective * material.specular.color;
                    }
                    else
                    {
                        pathSegments[idx].color *= (materialColor) * 1.0f;
                    }

                    pathSegments[idx].remainingBounces--;
                }
                // If there was no intersection, color the ray black.
                // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
                // used for opacity, in which case they can indicate "no opacity".
                // This can be useful for post-processing and image compositing.
            }
            else {
                pathSegments[idx].color = glm::vec3(0.0f);
                pathSegments[idx].remainingBounces = 0;
            }
        }
    }
}



// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
    int iter
    , int num_paths
    , ShadeableIntersection * shadeableIntersections
    , PathSegment * pathSegments
    , Material * materials
    )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) { // if the intersection exists...
            // Set up the RNG
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance);
            }
            // Otherwise, do some pseudo-lighting computation. This is actually more
            // like what you would expect from shading in a rasterizer like OpenGL.
            else {
                float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
                pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
                pathSegments[idx].color *= u01(rng); // apply some noise because why not
            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
        }
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        index = iterationPaths[index].pixelIndex;
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

// Add the current iteration's output to the current image
__global__ void partialGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        //index = iterationPaths[index].pixelIndex;
        if (iterationPaths[index].remainingBounces == 0)
        {
            PathSegment iterationPath = iterationPaths[index];
            image[iterationPath.pixelIndex] += iterationPath.color;
        }
    }
}

/**
* Wrapper for the __global__ call that sets up the kernel calls and does a ton
* of memory management
*/
void pathtrace(uchar4 *pbo, 
               int frame, 
               int iter, 
               float focalLength, 
               float dofAngle, 
               bool cacherays, 
               bool antialias, 
               float softness, 
               bool enableSss,
               bool testingmode,
               bool compaction,
               bool enablekd,
               bool vizkd) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 32;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing
    
    hipEvent_t startGenRayFromCam, stopGenRayFromCam;
    hipEvent_t startPathTraceOneBounce, stopPathTraceOneBounce;
    hipEvent_t startShadeMaterial, stopShadeMaterial;
    float millisecondsGenRayFromCam = 0.0f;
    float millisecondsPathTraceOneBounce = 0.0f;
    float millisecondsShadeMaterial = 0.0f;

    float ms1 = 0.0;
    float ms2 = 0.0;
    float ms3 = 0.0;
    /*
    if (testingmode)
    {
        hipEventCreate(&startGenRayFromCam); hipEventCreate(&stopGenRayFromCam); hipEventRecord(startGenRayFromCam);
    }
    */
    // cache rays
    if (cacherays)
    {
        if (iter == 1)
        {
            generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> >(cam, iter, traceDepth, dev_paths_cache, focalLength, dofAngle, antialias);
            checkCUDAError("generate camera ray");
        }
        hipMemcpy(dev_paths, dev_paths_cache, pixelcount*sizeof(PathSegment), hipMemcpyDeviceToDevice);
    }
    else
    {
        generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> >(cam, iter, traceDepth, dev_paths, focalLength, dofAngle, antialias);
        checkCUDAError("generate camera ray");
    }
    /*
    if (testingmode)
    {
        hipEventRecord(stopGenRayFromCam); hipEventSynchronize(stopGenRayFromCam);
        ms1 = 0; 
        hipEventElapsedTime(&ms1, startGenRayFromCam, stopGenRayFromCam);
        //printf("\ngenerateRayFromCamera time = %f", ms1);
        millisecondsGenRayFromCam = ms1;
        hipEventDestroy(startGenRayFromCam);
        hipEventDestroy(stopGenRayFromCam);
    }
    */
    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;
    int num_paths_temp = num_paths;
    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    //PathSegment* paths;
    //hipMalloc(&paths, sizeof(PathSegment)*pixelcount);
    //hipMemcpy(paths, dev_paths, sizeof(PathSegment)*pixelcount);
    
    bool iterationComplete = false;
    while (!iterationComplete) {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        
        if (testingmode)
        {
            hipEventCreate(&startPathTraceOneBounce); hipEventCreate(&stopPathTraceOneBounce); hipEventRecord(startPathTraceOneBounce);
        }
        
        if (enablekd == false)
        {
            pathTraceOneBounce << <numblocksPathSegmentTracing, blockSize1d >> > (
                depth
                , iter
                , num_paths
                , dev_paths
                , dev_geoms
                , hst_scene->geoms.size()
                , dev_materials
                , hst_scene->materials.size()
                , dev_intersections
                , softness
                , hst_scene->obj_numshapes
                , obj_numpolyverts
                , obj_verts
                , obj_norms
                , obj_texts
                , obj_polyoffsets
                , obj_polysidxflat
                , obj_polysbboxes
                , hst_scene->polyidxcount
                //, obj_RGB
                //, obj_SPECEX
                //, obj_SPECRGB
                //, obj_REFL
                //, obj_REFR
                //, obj_REFRIOR
                , obj_materialOffsets
                , hst_scene->hasObj);
            checkCUDAError("trace one bounce");
            ///*
            //printf("numNodes = %d\n", hst_scene->numNodes);
            //printf("numTriangles = %d\n", hst_scene->numTriangles);

        }
        else
        {
            if (vizkd)
            {
                pathTraceOneBounceKDbareBoxes << <numblocksPathSegmentTracing, blockSize1d >> > (
                    depth
                    , iter
                    , num_paths
                    , dev_paths
                    , dev_geoms
                    , hst_scene->geoms.size()
                    , dev_materials
                    , hst_scene->materials.size()
                    , dev_intersections
                    , softness
                    , kd_trianglesBare
                    , hst_scene->numTriangles
                    , kd_nodesBare
                    , hst_scene->numNodes
                    , obj_materialOffsets
                    , hst_scene->hasObj);
                checkCUDAError("trace one bounce kd");
                //hipEventQuery(0);
            }
            else
            {
                pathTraceOneBounceKDbare << <numblocksPathSegmentTracing, blockSize1d >> > (
                    depth
                    , iter
                    , num_paths
                    , dev_paths
                    , dev_geoms
                    , hst_scene->geoms.size()
                    , dev_materials
                    , hst_scene->materials.size()
                    , dev_intersections
                    , softness
                    , kd_trianglesBare
                    , hst_scene->numTriangles
                    , kd_nodesBare
                    , hst_scene->numNodes
                    , obj_materialOffsets
                    , hst_scene->hasObj);
                checkCUDAError("trace one bounce kd");
                //hipEventQuery(0);
            }
        }

        hipDeviceSynchronize();
        depth++;

        if (testingmode)
        {
            hipEventRecord(stopPathTraceOneBounce); hipEventSynchronize(stopPathTraceOneBounce);
            ms2 = 0;
            hipEventElapsedTime(&ms2, startPathTraceOneBounce, stopPathTraceOneBounce);
            //printf("\npath trace time = %f", ms2);
            millisecondsPathTraceOneBounce += ms2;
            hipEventDestroy(startPathTraceOneBounce);
            hipEventDestroy(stopPathTraceOneBounce);
        }

        /*
        if (testingmode)
        {
            hipEventCreate(&startShadeMaterial); hipEventCreate(&stopShadeMaterial); hipEventRecord(startShadeMaterial);
        }
        */
        shadeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials,
            enableSss
            );
        /*
        if (testingmode)
        {
            hipEventRecord(stopShadeMaterial); hipEventSynchronize(stopShadeMaterial);
            ms3 = 0;
            millisecondsShadeMaterial = 0;
            hipEventElapsedTime(&ms3, startGenRayFromCam, stopShadeMaterial);
            //printf("\ngenerateRayFromCamera time = %f", ms3);
            millisecondsShadeMaterial += ms3;
            hipEventDestroy(startGenRayFromCam);
            hipEventDestroy(stopShadeMaterial);
        }
        */
        //if (depth > 2)
        //if (num_paths <= 0)
        //    iterationComplete = true; // TODO: should be based off stream compaction results.

        if (compaction)
        {
            dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
            partialGather << <numBlocksPixels, blockSize1d >> >(num_paths, dev_image, dev_paths);
        }

        if (compaction)
        {
            thrust::device_ptr<PathSegment> thrust_paths(dev_paths);
            thrust::device_ptr<PathSegment> P = thrust::remove_if(thrust_paths, thrust_paths + num_paths, is_zero_bounce());
            num_paths_temp = P - thrust_paths;
            num_paths = num_paths_temp;
        }
        
        // with initial sort 00:45.32 without initial sort 00:49.13 with continous sort 01:11:73
        
        // after first hit
        if (iter == 2)
        {
            //printf("\nSORTING\n");
            thrust::device_ptr<PathSegment> thrust_paths2(dev_paths);
            thrust::sort(thrust_paths2, thrust_paths2 + num_paths);
            thrust::device_ptr<ShadeableIntersection> thrust_intersections(dev_intersections);
            thrust::sort(thrust_intersections, thrust_intersections + num_paths);
        }
        
        // stop if numpaths is 0 or depth > 8 when testing without compaction
        if (num_paths <= 0 || depth > 7)
            iterationComplete = true; // TODO: should be based off stream compaction results.  
    }
    
    if (testingmode)
    {
        //printf("\n\n-------- average times --------");
        //printf("\ngenerateRayFromCamera time = %f", millisecondsGenRayFromCam);
        printf("\n   pathTraceOneBounce time = %f", millisecondsPathTraceOneBounce);// / iter);
        //printf("\n        shadeMaterial time = %f\n", millisecondsShadeMaterial / iter);

        //printf("\n[%f, %f, %f], ", millisecondsGenRayFromCam, 
        //                           millisecondsPathTraceOneBounce / iter, 
        //                           millisecondsShadeMaterial / iter);
    }
    

    if (!compaction)
    {
        // Assemble this iteration and apply it to the image
        dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
        finalGather << <numBlocksPixels, blockSize1d >> >(num_paths, dev_image, dev_paths);
    }

    /*
    //printf("\ndev_paths %d\n", dev_paths[0].color.r);
    thrust::device_ptr<PathSegment> thrust_paths(dev_paths);
    thrust::device_ptr<PathSegment> P = thrust::remove_if(thrust_paths, thrust_paths + num_paths, is_zero_bounce());
    num_paths_temp = P - thrust_paths;
    num_paths -= num_paths_temp;
    */
    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    //if (iter == 25) 
    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> >(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}


